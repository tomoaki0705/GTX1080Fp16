
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

extern "C" void
launchCudaProcessHalf(dim3 grid, dim3 block, int sbytes,
					short *gain, float *imageInput, float *imageOutput);

extern "C" void
launchCudaProcessFloat(dim3 grid, dim3 block, int sbytes,
					float *gain, float *imageInput, float *imageOutput);

__global__ void
cudaProcessHalf(half2 *dst, half2 *gain, half2 *src, int imgW)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x*bw + tx;
	int y = blockIdx.y*bh + ty;
	int px = (y*imgW+x);

	half2 g = gain[px];
	half2 i = src[px];

	dst[px] = __hmul2(g, i);
}
